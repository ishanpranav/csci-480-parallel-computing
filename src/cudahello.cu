// Author: Mark Ebersole � NVIDIA Corporation
// Source: https://developer.nvidia.com/cuda-education
// Modified and adapted by Ishan Pranav


#include <hip/hip_runtime.h>
#include <stdio.h>

using namespace std;

__device__ const char *MESSAGE = "HELLO WORLD!";

const char MESSAGE_LENGTH = 12;

__global__ void hello()
{
    printf("%c\n", MESSAGE[threadIdx.x % MESSAGE_LENGTH]);
}

int main()
{
    int threads = MESSAGE_LENGTH;
    int blocks = 1;

    hello<<<blocks, threads>>>();
    hipDeviceSynchronize();

    return 0;
}
