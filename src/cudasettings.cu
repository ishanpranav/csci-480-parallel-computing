#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

int main()
{
    hipError_t error;
    hipDeviceProp_t device;
    int deviceCount = 0;

    error = hipGetDeviceCount(&deviceCount);

    if (error != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(error));
        
        return -1;
    }

    printf("Number of devices: %d\n", deviceCount);

    for (int i = 0; i < deviceCount; i++)
    {
        error = hipGetDeviceProperties(&device, i);

        if (error != hipSuccess)
        {
            printf("Error: %s\n", hipGetErrorString(error));
            
            return -1;
        }
        
        printf("\nDevice %d:\n", i);
        printf("name: %s\n", device.name);
        printf("Compute capability %d.%d\n", device.major, device.minor);
        printf("total global memory(KB): %ld\n", device.totalGlobalMem / 1024);
        printf("shared mem per block: %d\n", device.sharedMemPerBlock);
        printf("regs per block: %d\n", device.regsPerBlock);
        printf("warp size: %d\n", device.warpSize);
        printf("max threads per block: %d\n", device.maxThreadsPerBlock);
        printf("max thread dim x:%d y:%d z:%d\n", device.maxThreadsDim[0], device.maxThreadsDim[1], device.maxThreadsDim[2]);
        printf("max grid size x:%d y:%d z:%d\n", device.maxGridSize[0], device.maxGridSize[1], device.maxGridSize[2]);
        printf("clock rate(KHz): %d\n", device.clockRate);
        printf("total constant memory (bytes): %ld\n", device.totalConstMem);
        printf("multiprocessor count %d\n", device.multiProcessorCount);
        printf("integrated: %d\n", device.integrated);
        printf("async engine count: %d\n", device.asyncEngineCount);
        printf("memory bus width: %d\n", device.memoryBusWidth);
        printf("memory clock rate (KHz): %d\n", device.memoryClockRate);
        printf("L2 cache size (bytes): %d\n", device.l2CacheSize);
        printf("max threads per SM: %d\n", device.maxThreadsPerMultiProcessor);
    }

    return 1;
}
